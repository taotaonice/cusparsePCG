#include "hip/hip_runtime.h"
#include "cuda_pcg.h"
#include <ctime>

CUDA_pcg::CUDA_pcg()
{
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "hipsparseCreate failture  !!!" << std::endl;
        return ;
    }
    /* create and setup matrix descriptor */
    cusparseStatus = hipsparseCreateMatDescr(&descrA);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseCreateMatDescrA failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseCreateMatDescr(&descrB);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseCreateMatDescrB failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseCreateMatDescr(&descrC);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseCreateMatDescrC failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseCreateMatDescr(&descrL);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseCreateMatDescrL failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseCreateMatDescr(&descrH);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseCreateMatDescrH failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatTypeA failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatTypeB failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatTypeC failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatTypeL failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatType(descrH, HIPSPARSE_MATRIX_TYPE_GENERAL);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatTypeH failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatIndexBaseA failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatIndexBaseB failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatIndexBaseC failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatIndexBaseL failture  !!!" << std::endl;
        return ;
    }
    cusparseStatus = hipsparseSetMatIndexBase(descrH, HIPSPARSE_INDEX_BASE_ZERO);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout << "cusparseSetMatIndexBaseH failture  !!!" << std::endl;
        return ;
    }
}

CUDA_pcg::~CUDA_pcg()
{
    /* Destroy contexts */
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
}

__global__ void cuSetValue(int* value, int data_len)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < data_len)
    {
        value[tid] = tid;
    }
}

__global__ void cuCopyCfsToR(csrformatstruct* cfs, int* rowInd, int data_len)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < data_len)
    {
        rowInd[tid] = cfs[tid].row;
    }
}

__global__ void cuCopyRcvToCfs(csrformatstruct* cfs, int* rowInd, int* colInd, float* value, int data_len)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < data_len)
    {
        cfs[tid].row = rowInd[tid];
        cfs[tid].col = colInd[tid];
        cfs[tid].value = value[tid];
    }
}

__global__ void cuCopyCfsToRcv(csrformatstruct* cfs, int* rowInd, int* colInd, float* value, int data_len)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < data_len)
    {
        rowInd[tid] = cfs[tid].row;
        colInd[tid] = cfs[tid].col;
        value[tid] = cfs[tid].value;
    }
}

// sum(C, 2) --- C = A+A';
__global__ void cuSumA2(float* D, int* row, int* col, float* A, int length)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if(idx < length)
    {
        atomicAdd(&D[row[idx]], A[idx]);
        atomicAdd(&D[col[idx]], A[idx]);

//        if(row[idx] == col[idx])
//        {
//            atomicAdd(&D[row[idx]], -2 * A[idx]);
//        }
    }
}

__global__ void cuSetCarValD(float* D, float* csrValDiagAllCon, float lambda, int nz)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx < nz)
    {
        D[idx] = lambda * csrValDiagAllCon[idx];
    }
}

void CUDA_pcg::setSortedCsrFormat(int* rowInd, int* colInd, float* value, int nnz,
                                  int* sortedRowInd, int* sortedColInd, float* sortedValue)
{
//    TimingGPU tg;

    int threads = 256;
    int blocks = nnz /threads + ((nnz % threads) ? 1:0);

    csrformatstruct* cfs = NULL;
    hipMalloc((void**)&cfs, sizeof(csrformatstruct) * nnz);

    cuCopyRcvToCfs<<<blocks, threads>>>(cfs, rowInd, colInd, value, nnz);

    int* tmp_rowInd = NULL;
    hipMalloc((void**)&tmp_rowInd, sizeof(int) * nnz);
    hipMemcpy(tmp_rowInd, rowInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);

    //    tg.StartCounter();
    thrust::device_ptr<int> dev_row_ptr(tmp_rowInd);
    thrust::device_ptr<csrformatstruct> dev_csf_ptr(cfs);

//    tg.StartCounter();
    thrust::sort_by_key(dev_row_ptr, dev_row_ptr + nnz, dev_csf_ptr);
//    std::cout << "col thrust_time = " << tg.GetCounter() << " ms" << std::endl;

    csrformatstruct* first_sorted_cfs = thrust::raw_pointer_cast(dev_csf_ptr);

    cuCopyCfsToRcv<<<blocks, threads>>>(first_sorted_cfs, sortedRowInd, sortedColInd, sortedValue, nnz);

    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %d %s.\n", err, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }

    hipFree(cfs);
    hipFree(tmp_rowInd);
}


void CUDA_pcg::setSortedCsrFormatTwice(int* rowInd, int* colInd, float* value, int nnz,
                                  int* sortedRowInd, int* sortedColInd, float* sortedValue)
{
    int threads = 256;
    int blocks = nnz /threads + ((nnz % threads) ? 1:0);

    csrformatstruct* cfs = NULL;
    hipMalloc((void**)&cfs, sizeof(csrformatstruct) * nnz);

    cuCopyRcvToCfs<<<blocks, threads>>>(cfs, rowInd, colInd, value, nnz);

    int* tmp_colInd = NULL;
    hipMalloc((void**)&tmp_colInd, sizeof(int) * nnz);
    hipMemcpy(tmp_colInd, colInd, sizeof(int) * nnz, hipMemcpyDeviceToDevice);

    thrust::device_ptr<int> dev_col_ptr(tmp_colInd);
    thrust::device_ptr<csrformatstruct> dev_csf_ptr(cfs);

    thrust::sort_by_key(dev_col_ptr, dev_col_ptr + nnz, dev_csf_ptr);

    csrformatstruct* first_sorted_cfs = thrust::raw_pointer_cast(dev_csf_ptr);

    int* first_sorted_row = NULL;
    hipMalloc((void**)&first_sorted_row, sizeof(int) * nnz);
    cuCopyCfsToR<<<blocks, threads>>>(first_sorted_cfs, first_sorted_row, nnz);

    thrust::device_ptr<int> thrust_first_sorted_row(first_sorted_row);

    thrust::sort_by_key(thrust_first_sorted_row, thrust_first_sorted_row + nnz, dev_csf_ptr);


    csrformatstruct* second_sorted_cfs = thrust::raw_pointer_cast(dev_csf_ptr);

    cuCopyCfsToRcv<<<blocks, threads>>>(second_sorted_cfs, sortedRowInd, sortedColInd, sortedValue, nnz);

    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %d %s.\n", err, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }

    hipFree(cfs);
    hipFree(first_sorted_row);
    hipFree(tmp_colInd);
}



void CUDA_pcg::pcg(int* cooRowIndexHostPtr, int* cooColIndexHostPtr,
         float* cooValHostPtr, int M, int N, int nz,
         float* x, float *rhs, float *csrValDiagAllCon)
{
//    TimingGPU tg;
    int threads = 256;
    int blocks = M / threads + ((M % threads) ? 1:0);

    int* sortedRowIndA = NULL;
    int* sortedColIndA = NULL;
    float* sortedValueA = NULL;

    int n = M;
    assert(M == N);
    int nnz = nz;

    /* allocate GPU memory and copy the matrix and vectors into it */
    cudaStat1 = hipMalloc((void**)&sortedRowIndA, nnz*sizeof(sortedRowIndA[0]));
    cudaStat2 = hipMalloc((void**)&sortedColIndA, nnz*sizeof(sortedColIndA[0]));
    cudaStat3 = hipMalloc((void**)&sortedValueA, nnz*sizeof(sortedValueA[0]));

    if ((cudaStat1 != hipSuccess) ||
            (cudaStat2 != hipSuccess) ||
            (cudaStat3 != hipSuccess))
    {
        std::cout << "malloc 2 failture  !!!" << std::endl;
        return ;
    }

    cudaStat1 = hipMemcpy(sortedRowIndA, cooRowIndexHostPtr,
                           (size_t)(nnz*sizeof(sortedRowIndA[0])),
            hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(sortedColIndA, cooColIndexHostPtr,
                           (size_t)(nnz*sizeof(sortedColIndA[0])),
            hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(sortedValueA, cooValHostPtr,
                           (size_t)(nnz*sizeof(sortedValueA[0])),
            hipMemcpyHostToDevice);

    if ((cudaStat1 != hipSuccess) ||
            (cudaStat2 != hipSuccess) ||
            (cudaStat3 != hipSuccess))
    {
        std::cout << "hipMemcpy 1 failture  !!!" << std::endl;
        return ;
    }

    /* get sorted col index and sorted key for ATranspose*/
//    std::cout << "get sorted row index and sorted key for AT: " << std::endl;
//    tg.StartCounter();

    int* sortedRowIndAT = NULL;
    int* sortedColIndAT = NULL;
    float* sortedValueAT = NULL;

    hipMalloc((void**)&sortedRowIndAT, sizeof(int) * nnz);
    hipMalloc((void**)&sortedColIndAT, sizeof(int) * nnz);
    hipMalloc((void**)&sortedValueAT, sizeof(float) * nnz);


    setSortedCsrFormat(sortedColIndA, sortedRowIndA, sortedValueA, nnz,
                       sortedRowIndAT, sortedColIndAT, sortedValueAT);
//    std::cout << "sort time: " << tg.GetCounter() << std::endl;

//    // validate
//    int* h_sortedRowIndAT = new int[nnz];
//    int* h_sortedColIndAT = new int[nnz];
//    float* h_sortedValueAT = new float[nnz];

//    hipMemcpy(h_sortedRowIndAT, sortedRowIndAT, sizeof(int) * nnz, hipMemcpyDeviceToHost);
//    hipMemcpy(h_sortedColIndAT, sortedColIndAT, sizeof(int) * nnz, hipMemcpyDeviceToHost);
//    hipMemcpy(h_sortedValueAT, sortedValueAT, sizeof(float) * nnz, hipMemcpyDeviceToHost);

    // validate
//    std::cout << "h_sortedRowIndAT: " << std::endl;
//    for(int i = 0; i < 200; ++i)
//    {
//        std::cout << h_sortedRowIndAT[i] << "  " << h_sortedColIndAT[i] << "  " << h_sortedValueAT[i] << std::endl;
//    }

//    for(int i = 0; i < nnz; ++i)
//    {
//        std::cout << h_sortedRowIndAT[i] << "  " << h_sortedColIndAT[i] << "  " << h_sortedValueAT[i] << std::endl;
//    }

//    for(int i = 0; i < nnz; ++i)
//    {
//        std::cout << h_sortedValueAT[i] << std::endl;
//    }

    /* exercise conversion routines (convert matrix from COO 2 CSR format) */

        /* set csrRowPtrA */
//        tg.StartCounter();

        int* csrRowPtrA = NULL;
        cudaStat1 = hipMalloc((void**)&csrRowPtrA, (n + 1)*sizeof(csrRowPtrA[0]));
        if (cudaStat1 != hipSuccess)
        {
            std::cout << "hipMalloc csrRowPtrA failture  !!!" << std::endl;
            return ;
        }

        cusparseStatus = hipsparseXcoo2csr(cusparseHandle,
                                          sortedRowIndA,
                                          nnz,
                                          n,
                                          csrRowPtrA,
                                          HIPSPARSE_INDEX_BASE_ZERO);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcoo2csr csrRowPtrA failture  !!!" << std::endl;
            return ;
        }
//        std::cout << "set csrRowPtrA time = " << tg.GetCounter() << " ms" << std::endl;

//        // test  csrRowPtrHostA
//        int* csrRowPtrHostA = new int[n + 1];
//        hipError_t(hipMemcpy(csrRowPtrHostA, csrRowPtrA, (n + 1)*sizeof(csrRowPtrA[0]), hipMemcpyDeviceToHost));
//        std::cout << "csrRowPtrHostA: " << std::endl;
//        for(int i = 0; i < n + 1; ++i)
//        {
//            std::cout << csrRowPtrHostA[i] << std::endl;
//        }

        /* set csrRowPtrAT */
//        tg.StartCounter();

        int* csrRowPtrAT = NULL;
        cudaStat1 = hipMalloc((void**)&csrRowPtrAT, (n + 1)*sizeof(csrRowPtrA[0]));
        if (cudaStat2 != hipSuccess)
        {
            std::cout << "hipMalloc csrRowPtrAT failture  !!!" << std::endl;
            return ;
        }
        cusparseStatus = hipsparseXcoo2csr(cusparseHandle,
                                          sortedRowIndAT,
                                          nnz,
                                          n,
                                          csrRowPtrAT,
                                          HIPSPARSE_INDEX_BASE_ZERO);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcoo2csr csrColPtr failture  !!!" << std::endl;
            return ;
        }
//        std::cout << "set csrRowPtrAT time = " << tg.GetCounter() << " ms" << std::endl;

//        // test  csrRowPtrHostA
//        int* csrRowPtrHostAT = new int[n + 1];
//        hipError_t(hipMemcpy(csrRowPtrHostAT, csrRowPtrAT, (n + 1)*sizeof(csrRowPtrAT[0]), hipMemcpyDeviceToHost));
//        std::cout << "csrRowPtrHostAT: " << std::endl;
//        for(int i = 0; i < n + 1; ++i)
//        {
//            std::cout << csrRowPtrHostAT[i] << std::endl;
//        }

        /* A = A + A' */
//        tg.StartCounter();

        int baseC, nnzC;
        int *nnzTotalDevHostPtr = &nnzC;
        hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST);
        int* csrRowPtrC = NULL;
        hipError_t(hipMalloc((void**)&csrRowPtrC, sizeof(int) * (M + 1)));

        cusparseStatus = hipsparseXcsrgeamNnz(cusparseHandle, M, N,
                                             descrA, nz,
                                             csrRowPtrA, sortedColIndA,
                                             descrB, nz,
                                             csrRowPtrAT, sortedColIndAT,
                                             descrC,
                                             csrRowPtrC, nnzTotalDevHostPtr);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcsrgeamNnz csrRowPtrC failture 1 !!!" << std::endl;
            return ;
        }

        if(NULL != nnzTotalDevHostPtr)
        {
            nnzC = *nnzTotalDevHostPtr;
        }
        else
        {
            hipMemcpy(&nnzC, csrRowPtrC + M, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
            nnzC -= baseC;
        }

        const float A_alpha = 1.0;
        const float A_beta = 1.0;

        float* csrValC = NULL;
        int* csrColIndC = NULL;
        hipError_t(hipMalloc((void**)&csrValC, sizeof(float) * nnzC));
        hipError_t(hipMalloc((void**)&csrColIndC, sizeof(int) * nnzC));

        cusparseStatus = hipsparseScsrgeam(cusparseHandle, M, N,
                         &A_alpha,
                         descrA, nz,
                         sortedValueA, csrRowPtrA, sortedColIndA,
                         &A_beta,
                         descrB, nz,
                         sortedValueAT, csrRowPtrAT, sortedColIndAT,
                         descrC,
                         csrValC, csrRowPtrC, csrColIndC);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseScsrgeam csrValC failture  !!!" << std::endl;
            return ;
        }
//        std::cout << "A + A' time = " << tg.GetCounter() << std::endl;

//        // validate csrRowPtrC
//        int* h_csrRowPtrC = new int[M + 1];
//        int* h_csrColIndC = new int[nnzC];
//        float* h_csrValC = new float[nnzC];

//        hipMemcpy(h_csrRowPtrC, csrRowPtrC, sizeof(int) * (M + 1), hipMemcpyDeviceToHost);
//        hipMemcpy(h_csrColIndC, csrColIndC, sizeof(int) * nnzC, hipMemcpyDeviceToHost);
//        hipMemcpy(h_csrValC, csrValC, sizeof(float) * nnzC, hipMemcpyDeviceToHost);

//        std::cout << "h_csrRowPtrC: " << std::endl;
//        for(int i = 0; i < M + 1; ++i)
//        {
//            std::cout << h_csrRowPtrC[i] << std::endl;
//        }

//        std::cout << "h_csrColIndC h_csrValC: " << std::endl;
//        for(int i = 0; i < 500; ++i)
//        {
//            std::cout << h_csrColIndC[i] << "  " << h_csrValC[i] << std::endl;
//        }

//        std::cout << "h_csrColIndC h_csrValC: " << std::endl;
//        for(int i = nnzC - 500; i < nnzC; ++i)
//        {
//            std::cout << h_csrColIndC[i] << "  " << h_csrValC[i] << std::endl;
//        }
        ///////////////////////////////////////////
        ///
        ///
        ///             A + A'
        ///
        ///
        /// ///////////////////////////////////////////


//        tg.StartCounter();
        int* csrRowIndC = NULL;
        hipMalloc((void**)&csrRowIndC, sizeof(int) * nnzC);

        cusparseStatus = hipsparseXcsr2coo(cusparseHandle,
                                          csrRowPtrC,
                                          nnzC,
                                          M,
                                          csrRowIndC,
                                          HIPSPARSE_INDEX_BASE_ZERO);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseScsrgeam csrValC failture  !!!" << std::endl;
            return ;
        }
//        std::cout << "set A + A' row time " << tg.GetCounter() << std::endl;

//        int* h_csrRowIndC = new int[nnzC];
//        hipMemcpy(h_csrRowIndC, csrRowIndC, sizeof(int) * nnzC, hipMemcpyDeviceToHost);
//        std::cout << "h_csrRowIndC: " << std::endl;
//        for(int i = 0; i < nnzC; ++i)
//        {
//            std::cout << h_csrRowIndC[i] << std::endl;
//        }


        // set sum(A, 2) and add lambda * spdiags(all_constraints, 0, n_pick, n_pick)
//        tg.StartCounter();
        float* csrValD  = NULL;
        hipMalloc((void**)&csrValD, sizeof(int) * M);

        //float lambda = 100.0f;
        // inital carValD value to lambda * spdiags(all_constraints, 0, n_pick, n_pick)

        cuSetCarValD<<<blocks, threads>>>(csrValD, csrValDiagAllCon, 1, M);

//        float* h_csrValD = new float[M];
//        hipMemcpy(h_csrValD, csrValD, sizeof(float) * M, hipMemcpyDeviceToHost);
//        std::cout << "h_csrValD: " << std::endl;
//        for(int i = 0; i < M; ++i)
//        {
//            std::cout << h_csrValD[i] << std::endl;
//        }


//        cuSumA2<<<blocks, threads>>>(csrValD, csrRowIndC, csrColIndC, csrValC, nz);
        int blocks_sumA = nz / threads + ((nz % threads) ? 1:0);
        cuSumA2<<<blocks_sumA, threads>>>(csrValD, sortedRowIndA, sortedColIndA, sortedValueA, nz);
//        std::cout << "set csrValD time = " << tg.GetCounter() << " ms" << std::endl;

//        float* h_csrValD = new float[M];
//        hipMemcpy(h_csrValD, csrValD, sizeof(float) * M, hipMemcpyDeviceToHost);
//        std::cout << "h_csrValD: " << std::endl;
//        for(int i = 0; i < M; ++i)
//        {
//            std::cout << h_csrValD[i] << std::endl;
//        }


        ///////////////////////////////////////////
        ///
        ///
        ///             set D
        ///
        ///
        /// ///////////////////////////////////////////

//        tg.StartCounter();
        int* csrRowPtrD = NULL;
        int* csrColIndD = NULL;
        hipMalloc((void**)&csrRowPtrD, sizeof(int) * (M + 1));
        hipMalloc((void**)&csrColIndD, sizeof(int) * M);

        int blocks_ptr = (M + 1) / threads + (((M + 1) % threads) ? 1:0);
        cuSetValue<<<blocks_ptr, threads>>>(csrRowPtrD, M + 1);
        cuSetValue<<<blocks, threads>>>(csrColIndD, M);

//        std::cout << "cuSetValue time = " << tg.GetCounter() << " ms" << std::endl;

//        cusparseStatus = hipsparseXcoo2csr(cusparseHandle,
//                                          csrRowIndC,
//                                          M,  // nnz
//                                          M,
//                                          csrRowPtrD,
//                                          HIPSPARSE_INDEX_BASE_ZERO);
//        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
//        {
//            std::cout << "hipsparseXcoo2csr csrRowPtrD failture  !!!" << std::endl;
//            return ;
//        }
//        std::cout << "set csrRowPtrD time = " << tg.GetCounter() << " ms" << std::endl;

//        int* h_csrRowPtrD = new int[M];
//        hipMemcpy(h_csrRowPtrD, csrRowPtrD, sizeof(int) * M, hipMemcpyDeviceToHost);
//        std::cout << "h_csrRowPtrD: " << std::endl;
//        for(int i = 0; i < M; ++i)
//        {
//            std::cout << h_csrRowPtrD[i] << std::endl;
//        }


        // L = D - A
//        tg.StartCounter();

        int* csrRowPtrL = NULL;
        int* csrColIndL = NULL;
        float* csrValL = NULL;
        int nnzL = 0;

        const float L_alpha = 1.0;
        const float L_beta = -1.0;

        int baseL;
        int *nnzTotalDevHostPtrL = &nnzL;
        hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST);
        hipError_t(hipMalloc((void**)&csrRowPtrL, sizeof(int) * (M + 1)));

        cusparseStatus = hipsparseXcsrgeamNnz(cusparseHandle, M, N,
                                             descrA, M,
                                             csrRowPtrD, csrColIndD,
                                             descrC, nnzC,
                                             csrRowPtrC, csrColIndC,
                                             descrL,
                                             csrRowPtrL, nnzTotalDevHostPtrL);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcsrgeamNnz csrRowPtrC failture 2 !!!" << std::endl;
            return ;
        }

        if(NULL != nnzTotalDevHostPtrL)
        {
            nnzL = *nnzTotalDevHostPtrL;
        }
        else
        {
            hipMemcpy(&nnzL, csrRowPtrL + M, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&baseL, csrRowPtrL, sizeof(int), hipMemcpyDeviceToHost);
            nnzL -= baseL;
        }

        hipError_t(hipMalloc((void**)&csrValL, sizeof(float) * nnzL));
        hipError_t(hipMalloc((void**)&csrColIndL, sizeof(int) * nnzL));

        cusparseStatus = hipsparseScsrgeam(cusparseHandle, M, N,
                                          &L_alpha,
                                          descrA, M,
                                          csrValD, csrRowPtrD, csrColIndD,
                                          &L_beta,
                                          descrC, nnzC,
                                          csrValC, csrRowPtrC, csrColIndC,
                                          descrL,
                                          csrValL, csrRowPtrL, csrColIndL);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseScsrgeam csrValC failture  !!!" << std::endl;
            return ;
        }
//        std::cout << "set L time = " << tg.GetCounter() << " ms" << std::endl;

//        // validate csrRowPtrL
//        int* h_csrRowPtrL = new int[M + 1];
//        int* h_csrColIndL = new int[nnzL];
//        float* h_csrValL = new float[nnzL];

//        hipMemcpy(h_csrRowPtrL, csrRowPtrL, sizeof(int) * (M + 1), hipMemcpyDeviceToHost);
//        hipMemcpy(h_csrColIndL, csrColIndL, sizeof(int) * nnzL, hipMemcpyDeviceToHost);
//        hipMemcpy(h_csrValL, csrValL, sizeof(float) * nnzL, hipMemcpyDeviceToHost);

//        // validate csrRowPtrH
//        int* h_csrRowPtrL = new int[M + 1];
//        int* h_csrColIndL = new int[nnzL];
//        float* h_csrValL = new float[nnzL];

//        hipMemcpy(h_csrRowPtrL, csrRowPtrL, sizeof(int) * (M + 1), hipMemcpyDeviceToHost);
//        hipMemcpy(h_csrColIndL, csrColIndL, sizeof(int) * nnzL, hipMemcpyDeviceToHost);
//        hipMemcpy(h_csrValL, csrValL, sizeof(float) * nnzL, hipMemcpyDeviceToHost);

//        std::cout << "h_csrRowPtrL: " << std::endl;
//        for(int i = 0; i < M + 1; ++i)
//        {
//            std::cout << h_csrRowPtrL[i] << std::endl;
//        }

//        std::cout << "h_csrColIndL h_csrValL: " << std::endl;
//        for(int i = 0; i < 500; ++i)
//        {
//            std::cout << h_csrColIndL[i] << "  " << h_csrValL[i] << std::endl;
//        }

//        std::cout << "h_csrColIndL h_csrValL: \n\n" << std::endl;

//        for(int i = nnzL - 500; i < nnzL; ++i)
//        {
//            std::cout << h_csrColIndL[i] << "  " << h_csrValL[i] << std::endl;
//        }



        /*******************************/
        /*******************************/
        /***********  pcg  *************/
        /*******************************/
        /*******************************/

        const float tol = 1e-2f;
        const int max_iter = 100;
        const float floatone = 1.0;
        const float floatzero = 0.0;

        float* d_x = NULL;
        float* d_r = NULL;
        float* d_p = NULL;
        float* d_omega = NULL;

        float r0, r1, alpha, beta;;
        int k;
        float dot, nalpha;

        hipError_t(hipMalloc((void **)&d_x, N*sizeof(float)));
        hipError_t(hipMalloc((void **)&d_r, N*sizeof(float)));
        hipError_t(hipMalloc((void **)&d_p, N*sizeof(float)));
        hipError_t(hipMalloc((void **)&d_omega, N*sizeof(float)));

        // validate rhs
    //    std::cout << "rhs: " << std::endl;
//        float* rhs = new float[N];
//        for(int i = 0; i < N; ++i)
//        {
//            rhs[i] = lambda * foreground[i];
//    //        std::cout << rhs[i] << std::endl;
//        }

        hipError_t(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
        hipError_t(hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice));


//        tg.StartCounter();

        k = 0;
        r0 = 0;
        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

        while (r1 > tol*tol && k <= max_iter)
        {
            k++;
            if (k == 1)
            {
                hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
            }
            else
            {
                beta = r1/r0;
                hipblasSscal(cublasHandle, N, &beta, d_p, 1);
                hipblasSaxpy(cublasHandle, N, &floatone, d_r, 1, d_p, 1) ;
            }

            hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnzL, &floatone,
                           descrL, csrValL, csrRowPtrL, csrColIndL, d_p, &floatzero, d_omega);

            hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &dot);
            alpha = r1/dot;
            hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
            nalpha = -alpha;
            hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
            r0 = r1;
            hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        }

        printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));
//        std::cout << "get x time = " << tg.GetCounter() << " ms" << std::endl;

        hipError_t(hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost));
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//        /* check result */
//        err = 0.0;
//        for (int i = 0; i < N; i++)
//        {
//            rsum = 0.0;

//            for (int j = h_csrRowPtrL[i]; j < h_csrRowPtrL[i+1]; j++)
//            {
//                rsum += h_csrValL[j] * x[h_csrColIndL[j]];
//            }

//            diff = fabs(rsum - rhs[i]);

//            if (diff > err)
//            {
//                err = diff;
//            }
//        }
//        printf("  Convergence Test: %s \n", (k <= max_iter) ? "OK" : "FAIL");
//        nErrors += (k > max_iter) ? 1 : 0;
//        qaerr1 = err;

//        std::cout << "x....\n" ;
//        for(int i = 0; i < N; ++i)
//        {
//            printf(" %f \n", x[i]);
//        }

        hipFree(sortedRowIndA);
        hipFree(sortedColIndA);
        hipFree(sortedValueA);

        hipFree(sortedRowIndAT);
        hipFree(sortedColIndAT);
        hipFree(sortedValueAT);

        hipFree(csrRowPtrA);
        hipFree(csrRowPtrAT);
        hipFree(csrRowPtrC);
        hipFree(csrValC);
        hipFree(csrColIndC);
        hipFree(csrRowIndC);
        hipFree(csrValD);
        hipFree(csrRowPtrD);
        hipFree(csrColIndD);
        hipFree(csrRowPtrL);
        hipFree(csrValL);
        hipFree(csrColIndL);

        hipFree(d_x);
        hipFree(d_r);
        hipFree(d_p);
        hipFree(d_omega);
}


void CUDA_pcg::cu_pcg(int* sortedRowIndA, int* sortedColIndA,
         float* sortedValueA, int M, int N, int nz,
         float* x, float *rhs, float *csrValDiagAllCon)
{
//*********************************************
//    A = A + A';
//    D = spdiags(sum(A, 2), 0, N, N);
//    L = D - A;
//*********************************************

//*********************************************
//*********************************************
//*********************************************
//*********************************************
    int threads = 256;
    int blocks = M / threads + ((M % threads) ? 1:0);

    int n = M;
    assert(M == N);
    int nnz = nz;

    setSortedCsrFormat(sortedRowIndA, sortedColIndA, sortedValueA, nnz,
                       sortedRowIndA, sortedColIndA, sortedValueA);


    int* sortedRowIndAT = NULL;
    int* sortedColIndAT = NULL;
    float* sortedValueAT = NULL;

    hipMalloc((void**)&sortedRowIndAT, sizeof(int) * nnz);
    hipMalloc((void**)&sortedColIndAT, sizeof(int) * nnz);
    hipMalloc((void**)&sortedValueAT, sizeof(float) * nnz);


    setSortedCsrFormat(sortedColIndA, sortedRowIndA, sortedValueA, nnz,
                       sortedRowIndAT, sortedColIndAT, sortedValueAT);

        /* set csrRowPtrA */

        int* csrRowPtrA = NULL;
        cudaStat1 = hipMalloc((void**)&csrRowPtrA, (n + 1)*sizeof(csrRowPtrA[0]));
        if (cudaStat1 != hipSuccess)
        {
            std::cout << "hipMalloc csrRowPtrA failture  !!!" << std::endl;
            return ;
        }

        cusparseStatus = hipsparseXcoo2csr(cusparseHandle,
                                          sortedRowIndA,
                                          nnz,
                                          n,
                                          csrRowPtrA,
                                          HIPSPARSE_INDEX_BASE_ZERO);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcoo2csr csrRowPtrA failture  !!!" << std::endl;
            return ;
        }

        /* set csrRowPtrAT */

        int* csrRowPtrAT = NULL;
        cudaStat1 = hipMalloc((void**)&csrRowPtrAT, (n + 1)*sizeof(csrRowPtrA[0]));
        if (cudaStat1 != hipSuccess)
        {
            std::cout << "hipMalloc csrRowPtrAT failture  !!!" << std::endl;
            return ;
        }
        cusparseStatus = hipsparseXcoo2csr(cusparseHandle,
                                          sortedRowIndAT,
                                          nnz,
                                          n,
                                          csrRowPtrAT,
                                          HIPSPARSE_INDEX_BASE_ZERO);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcoo2csr csrColPtr failture  !!!" << std::endl;
            return ;
        }

        /* A = A + A' */

        int baseC, nnzC;
        int *nnzTotalDevHostPtr = &nnzC;
        hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST);
        int* csrRowPtrC = NULL;
        hipError_t(hipMalloc((void**)&csrRowPtrC, sizeof(int) * (M + 1)));

        cusparseStatus = hipsparseXcsrgeamNnz(cusparseHandle, M, N,
                                             descrA, nz,
                                             csrRowPtrA, sortedColIndA,
                                             descrB, nz,
                                             csrRowPtrAT, sortedColIndAT,
                                             descrC,
                                             csrRowPtrC, nnzTotalDevHostPtr);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcsrgeamNnz csrRowPtrC failture 3 !!!" << std::endl;
            return ;
        }

        if(NULL != nnzTotalDevHostPtr)
        {
            nnzC = *nnzTotalDevHostPtr;
        }
        else
        {
            hipMemcpy(&nnzC, csrRowPtrC + M, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
            nnzC -= baseC;
        }

        const float A_alpha = 1.0;
        const float A_beta = 1.0;

        float* csrValC = NULL;
        int* csrColIndC = NULL;
        hipError_t(hipMalloc((void**)&csrValC, sizeof(float) * nnzC));
        hipError_t(hipMalloc((void**)&csrColIndC, sizeof(int) * nnzC));

        cusparseStatus = hipsparseScsrgeam(cusparseHandle, M, N,
                         &A_alpha,
                         descrA, nz,
                         sortedValueA, csrRowPtrA, sortedColIndA,
                         &A_beta,
                         descrB, nz,
                         sortedValueAT, csrRowPtrAT, sortedColIndAT,
                         descrC,
                         csrValC, csrRowPtrC, csrColIndC);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseScsrgeam csrValC failture  !!!" << std::endl;
            return ;
        }

        ///////////////////////////////////////////
        ///
        ///
        ///             C = A + A'
        ///
        ///
        /// ///////////////////////////////////////////

        int* csrRowIndC = NULL;
        hipMalloc((void**)&csrRowIndC, sizeof(int) * nnzC);

        cusparseStatus = hipsparseXcsr2coo(cusparseHandle,
                                          csrRowPtrC,
                                          nnzC,
                                          M,
                                          csrRowIndC,
                                          HIPSPARSE_INDEX_BASE_ZERO);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseScsrgeam csrValC failture  !!!" << std::endl;
            return ;
        }

        // set sum(A, 2) and add lambda * spdiags(all_constraints, 0, n_pick, n_pick)
        float* csrValD  = NULL;
        hipMalloc((void**)&csrValD, sizeof(int) * M);

        hipMemcpy(csrValD, csrValDiagAllCon, sizeof(float) * M, hipMemcpyDeviceToDevice);

        int blocks_sumA = nz / threads + ((nz % threads) ? 1:0);
        cuSumA2<<<blocks_sumA, threads>>>(csrValD, sortedRowIndA, sortedColIndA, sortedValueA, nz);

        ///////////////////////////////////////////
        ///
        ///
        ///             set D
        ///
        ///
        /// ///////////////////////////////////////////

        int* csrRowPtrD = NULL;
        int* csrColIndD = NULL;
        hipMalloc((void**)&csrRowPtrD, sizeof(int) * (M + 1));
        hipMalloc((void**)&csrColIndD, sizeof(int) * M);

        int blocks_ptr = (M + 1) / threads + (((M + 1) % threads) ? 1:0);
        cuSetValue<<<blocks_ptr, threads>>>(csrRowPtrD, M + 1);
        cuSetValue<<<blocks, threads>>>(csrColIndD, M);

        // L = D - C

        int* csrRowPtrL = NULL;
        int* csrColIndL = NULL;
        float* csrValL = NULL;
        int nnzL = 0;

        const float L_alpha = 1.0;
        const float L_beta = -1.0;

        int baseL;
        int *nnzTotalDevHostPtrL = &nnzL;
        hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST);
        hipError_t(hipMalloc((void**)&csrRowPtrL, sizeof(int) * (M + 1)));

        cusparseStatus = hipsparseXcsrgeamNnz(cusparseHandle, M, N,
                                             descrA, M,
                                             csrRowPtrD, csrColIndD,
                                             descrC, nnzC,
                                             csrRowPtrC, csrColIndC,
                                             descrL,
                                             csrRowPtrL, nnzTotalDevHostPtrL);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseXcsrgeamNnz csrRowPtrC failture 4 !!!" << std::endl;
            return ;
        }

        if(NULL != nnzTotalDevHostPtrL)
        {
            nnzL = *nnzTotalDevHostPtrL;
        }
        else
        {
            hipMemcpy(&nnzL, csrRowPtrL + M, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&baseL, csrRowPtrL, sizeof(int), hipMemcpyDeviceToHost);
            nnzL -= baseL;
        }

        hipError_t(hipMalloc((void**)&csrValL, sizeof(float) * nnzL));
        hipError_t(hipMalloc((void**)&csrColIndL, sizeof(int) * nnzL));

        cusparseStatus = hipsparseScsrgeam(cusparseHandle, M, N,
                                          &L_alpha,
                                          descrA, M,
                                          csrValD, csrRowPtrD, csrColIndD,
                                          &L_beta,
                                          descrC, nnzC,
                                          csrValC, csrRowPtrC, csrColIndC,
                                          descrL,
                                          csrValL, csrRowPtrL, csrColIndL);
        if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout << "hipsparseScsrgeam csrValC failture  !!!" << std::endl;
            return ;
        }
//*********************************************
//*********************************************
//*********************************************
//*********************************************


        /*******************************/
        /*******************************/
        /***********  pcg  *************/
        /*******************************/
        /*******************************/

        static float sum = 0;
        static int cnts = 0;
        long t1 = clock();

        const float tol = 1e-5f;
        const int max_iter = 100;
        const float floatone = 1.0;
        const float floatzero = 0.0;
        const float floatnone = -1.0;

        float* d_x = NULL;
        float* d_r = NULL;
        float* d_p = NULL;
        float* d_omega = NULL;

        float r0, r1, alpha;
        int k;
        float dot, nalpha;

        hipError_t(hipMalloc((void **)&d_x, N*sizeof(float)));
        hipError_t(hipMalloc((void **)&d_r, N*sizeof(float)));
        hipError_t(hipMalloc((void **)&d_p, N*sizeof(float)));
        hipError_t(hipMalloc((void **)&d_omega, N*sizeof(float)));

        hipError_t(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyDeviceToDevice));
//        hipError_t(hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice));

        hipError_t(hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyDeviceToDevice));


        // omega = A*X
        hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnzL, &floatone,
                       descrL, csrValL, csrRowPtrL, csrColIndL, d_x, &floatzero, d_omega);
        // r = b - omega
        hipblasSaxpy(cublasHandle, N, &floatnone, d_omega, 1, d_r, 1);
        // p = r
        hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
        // r0 = r' * r
        hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r0);

        k = 0;

        while (k <= max_iter)
        {
            k++;

            // Ap = A*p
            hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnzL, &floatone,
                           descrL, csrValL, csrRowPtrL, csrColIndL, d_p, &floatzero, d_omega);

            // dot = p' * Ap
            hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &dot);
            alpha = r0 / dot;
            hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
            nalpha = -alpha;
            hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
            //r0 = r1;
            hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
            if (r1 < tol*tol) break;
            // printf("r1: %f\n", r1);
            // p = r + (r1 / r0)*p
            hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
            alpha = (r1 / r0);
            hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_p, 1);
            r0 = r1;
        }

//        printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));
//        std::cout << "get x time = " << tg.GetCounter() << " ms" << std::endl;

        hipError_t(hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToDevice));

        long t2 = clock();
        cnts++;
        sum += (t2-t1)*1000.0/CLOCKS_PER_SEC;
        std::cout<<cnts<<" times  ave_time: "<< sum/cnts<<" ms"<<std::endl;

        ///////////////////////////////////////////////////////////////////////////////////
        hipFree(sortedRowIndAT);
        hipFree(sortedColIndAT);
        hipFree(sortedValueAT);

        hipFree(csrRowPtrA);
        hipFree(csrRowPtrAT);
        hipFree(csrRowPtrC);
        hipFree(csrValC);
        hipFree(csrColIndC);
        hipFree(csrRowIndC);
        hipFree(csrValD);
        hipFree(csrRowPtrD);
        hipFree(csrColIndD);
        hipFree(csrRowPtrL);
        hipFree(csrValL);
        hipFree(csrColIndL);

        hipFree(d_x);
        hipFree(d_r);
        hipFree(d_p);
        hipFree(d_omega);
}
